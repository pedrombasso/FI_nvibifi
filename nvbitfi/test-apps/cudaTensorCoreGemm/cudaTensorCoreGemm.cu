#include "hip/hip_runtime.h"
#include "/home/carol/radiation-benchmarks/src/cuda/common/include/device_vector.h"
#include <vector>
#include <iostream>
#include <hip/hip_fp16.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>
#include <random>
#include <algorithm>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "common_template_functions.h"
#include "Parameters.h"


// Externally configurable parameters.

#ifndef CPU_DEBUG
// Set this to 1 to verify the correctness of the GPU-computed matrix.
#define CPU_DEBUG 0
#endif

#ifndef SHARED_MEMORY_LIMIT_64K
// Set this to 0 to use more than 64 Kb of shared memory to cache data, to
// improve the performance of the computations on GPU.
// Note that you need a GPU that can have more than 64 Kb of shared memory
// per multiprocessor.
#define SHARED_MEMORY_LIMIT_64K 0
#endif

// GPU configuration.

#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define M 16
#define N 16
#define K 16

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// GEMM configuration.

#define M_TILES 256 //512 // 128 for 2k, 512 for 8k etc 
#define N_TILES 256 //512 //
#define K_TILES 256 //512 //


#define M_GLOBAL (M * M_TILES)
#define N_GLOBAL (N * N_TILES)
#define K_GLOBAL (K * K_TILES)

#define BLOCK_SIZE 32

#define C_LAYOUT wmma::mem_row_major

// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
// With only 64 Kb shared memory available, we can fit two 8-tile chunks of
// the A and B matrix data, that are 16 * 16 * 8 * 8 * 2 = 32 Kb each
// (i.e. two 8x8 arrays of tiles of 16x16 half-typed elements per CTA).
// But we cannot account the 8 Kb total skew overhead, without which the
// performance would be severely impacted. So we choose to reduce the chunk size
// in half, i.e. the amount of A and B matrix data we cache in shared memory.
// Accordingly, this doubles the number of outer iterations across the global K
// dimension, which only slightly impacts the performance.
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(half))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)



// The macro below is used to shift rows of the A matrix and columns of the B
// matrix in shared memory to minimize possible bank conflicts. Before
// performing the nvcuda::wmma::mma_sync operation, the warp must load the
// matrix data using the nvcuda::wmma::load_matrix_sync operation. Although the
// memory access pattern is not specified for that function, each lane in the
// warp can read one or multiple matrix elements from different matrix rows or
// columns. For shared memory, such access can result in bank conflicts if
// different rows / columns of the matrix map to the same bank. By shifting each
// row and column by a few bytes, we make sure that they map to different banks,
// thus reducing the number of possible bank conflicts. The number of 8 two-byte
// "half" elements is chosen as the minimum possible shift because we must keep
// each row and column 128-bit aligned, as required by
// nvcuda::wmma::load_matrix_sync.
#define SKEW_HALF 8



//generete gold 
#define GOLD 0

#define DMR 0

#define CHAR_CAST(x) (reinterpret_cast<char*>(x))

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

using namespace nvcuda;

__global__ void compute_gemm(const half *A, const half *B, const half *C,
        half *D, half alpha, half beta) {
    extern __shared__ half shmem[][CHUNK_K * K + SKEW_HALF];

    // Warp and lane identification.
    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    // Offset in shared memory from which the B matrix is stored.
    const size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

    // This pointer is used to access the C and D matrix tiles this warp computes.
    half *shmem_warp_tile_ptr = (half *) &shmem[0][0]
            + (warpId / 2) * SHMEM_STRIDE * K * 2+
            (warpId % 2) * SHMEM_OFFSET;

    // This pointer is used to stream the C and D matrices block-wide tile to and
    // from shared memory.
    half *shmem_warp_stream_ptr = (half *) &shmem[0][0]
            + warpId * SHMEM_STRIDE * K;

    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
    // each tile computation. Technically this is not generally correct (may
    // result in a loss of precision). Zero still needs to be specially handled
    // though.
    beta /= alpha;

    // Each CTA slides along the 128 x 128 tiles from the top left corner of the
    // matrix to the right and down, and selects the next tile to compute. Once
    // there's no such tile, all warps in this CTA exit.
    for (unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
        const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES)
                / N_TILES) * (BLOCK_COL_TILES);
        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES)
                % N_TILES;

        // Stop when there are no more D matrix tiles to compute in this CTA.
        if (block_tile_i >= M_TILES) {
            break;
        }

        // This warp's pointer to the C matrix data to copy memory from to shared
        // memory.
        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE
                + block_tile_j * N;
        const half *src_gmem_warp_stream_ptr = &C[gmem_idx];

        // Stream multiple C tiles to shared memory.
#pragma unroll
        for (int i = 0; i < K; i++) {
            typedef int4 copy_t;

            *((copy_t *) (shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId) =
                    *((copy_t *) (src_gmem_warp_stream_ptr
                            + GLOBAL_MEM_STRIDE * i) + laneId);
        }

        __syncthreads();

        // These fragments will accumulate the result of A and B matrix fragment
        // multiplications along the K_GLOBAL dimension.
        nvcuda::wmma::fragment < nvcuda::wmma::accumulator, M, N, K, half
                > c[WARP_COL_TILES][WARP_ROW_TILES];

        // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                const half *tile_ptr = shmem_warp_tile_ptr
                        + i * SHMEM_STRIDE * K + j * N;

                nvcuda::wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE,
                C_LAYOUT);
            }
        }

        __syncthreads();

        // Scale the C matrix.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++) {
                    c[i][j].x[t] *= beta;
                }
            }
        }

        // Select what warp copies what matrix to shared memory.
        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
        const half *warp_ptr =
                (warpId < 4) ? (&A[block_tile_i * M * K_GLOBAL] +
                M * K_GLOBAL * (warpId % 4) * 2) :
                                (&B[block_tile_j * N * K_GLOBAL] +
                                N * K_GLOBAL * (warpId % 4) * 2);

        // Go through the global K dimension by a fixed step at a time.
#pragma unroll
        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
            // Copy slices of the A and B matrices to shared memory.
            // The first half of the warps in the CTA copy the A matrix, the rest copy
            // the B matrix.
            size_t shmem_idx =
                    warpId < (WARPS_PER_BLOCK / 2) ?
                            (M * (warpId % (WARPS_PER_BLOCK / 2)) * 2) :
                            (N * (warpId % (WARPS_PER_BLOCK / 2)) * 2
                                    + shmem_idx_b_off);

            // First half of the warp copies the first row / column of the matrix,
            // the second half of the warp copies the next.
            int4 *lane_ptr = (int4 *) (warp_ptr + tile_k * K
                    + (laneId / CHUNK_COPY_LINE_LANES) * K_GLOBAL)
                    + (laneId % CHUNK_COPY_LINE_LANES);

            // Shift the second half of the warp to the next row / column in the
            // shared memory.
            shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

#pragma unroll
            for (int i = 0;
                    i < ((WARP_SIZE / 2) / CHUNK_COPY_LINES_PER_WARP) * 2;
                    i++) {
                // Copy 16 bytes dodasat once in each lane.
                *((int4 *) &shmem[shmem_idx][0]
                        + (laneId % CHUNK_COPY_LINE_LANES)) = *lane_ptr;

                // Advance the global memory pointer and the shared memory index.
                lane_ptr = (int4 *) ((half *) lane_ptr
                        + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP);
                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
            }

            __syncthreads();

            // Compute a grid of C matrix tiles in each warp.
#pragma unroll
            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
                nvcuda::wmma::fragment < nvcuda::wmma::matrix_a, M, N, K, half, nvcuda::wmma::row_major
                        > a[WARP_COL_TILES];
                nvcuda::wmma::fragment < nvcuda::wmma::matrix_b, M, N, K, half, nvcuda::wmma::col_major
                        > b[WARP_ROW_TILES];

#pragma unroll
                for (int i = 0; i < WARP_COL_TILES; i++) {
                    size_t shmem_idx_a = (warpId / 2) * M * 2 + (i * M);
                    const half *tile_ptr = &shmem[shmem_idx_a][k_step * K];

                    nvcuda::wmma::load_matrix_sync(a[i], tile_ptr,
                    K * CHUNK_K + SKEW_HALF);

#pragma unroll
                    for (int j = 0; j < WARP_ROW_TILES; j++) {
                        if (i == 0) {
                            // Load the B matrix fragment once, because it is going to be
                            // reused against the other A matrix fragments.
                            size_t shmem_idx_b = shmem_idx_b_off
                                    + (WARP_ROW_TILES * N) * (warpId % 2)
                                    + (j * N);
                            const half *tile_ptr = &shmem[shmem_idx_b][k_step
                                    * K];

                            nvcuda::wmma::load_matrix_sync(b[j], tile_ptr,
                            K * CHUNK_K + SKEW_HALF);
                        }

                        nvcuda::wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
                    }
                }
            }

            __syncthreads();
        }

        // Store the D fragments to shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                // Uniform, point-wise transformations of ALL fragment elements by ALL
                // threads in the warp are well-defined even though element indices
                // within fragment storage are not defined.
                for (int t = 0; t < c[i][j].num_elements; t++)
                    c[i][j].x[t] *= alpha;

                half *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * K
                        + j * N;

                nvcuda::wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE,
                C_LAYOUT);
            }
        }

        __syncthreads();

        // Now that shared memory contains all the D tiles, stream them to global
        // memory.
        half *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
        for (int i = 0; i < K; i++) {
            *((int4 *) (dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i)
                    + laneId) = *((int4 *) (shmem_warp_stream_ptr
                    + SHMEM_STRIDE * i) + laneId);
        }

        __syncthreads();
    }
}




__host__ void generate_input_matrices(std::vector<half>& a_vector,
        std::vector<half>& b_vector) {

    std::random_device rd; //Will be used to obtain a seed for the random number engine
    std::mt19937 gen(rd()); //Standard mersenne_twister_engine seeded with rd()
    std::uniform_real_distribution<float> dis(0.1, 1.0);
    
    a_vector.resize(M_GLOBAL * M_GLOBAL);
    b_vector.resize(M_GLOBAL * M_GLOBAL);
    

    for (int i = 0; i < M_GLOBAL * M_GLOBAL; i++) {
        a_vector[i]=2.0; //half(dis(gen));
        b_vector[i]=2.0; //half(dis(gen));

    }    
       
}

int main(int argc, char **argv){
    constexpr auto n = M_GLOBAL;
    constexpr auto size = n * n;
    std::cout << "Size " << n << " elements " << size << std::endl;

    //host inputs
    std::vector<half> a_host(size, 0), b_host(size, 0), c_host(size, 0), d_host(size, 0), gold_host(size,0), relError(size, 0), relMinMax(2,0);    
    generate_input_matrices (a_host, b_host);


    //device matrices
    rad::DeviceVector<half> a = a_host;
    rad::DeviceVector<half> b = b_host;
    rad::DeviceVector<half> c_s = c_host;


    rad::DeviceVector<half> c_h = c_host;
    rad::DeviceVector<half> d_h = d_host;


    rad::DeviceVector<half> relErrorDevice = d_host;
    rad::DeviceVector<half> relMinMaxDevice = relMinMax;

    hipEvent_t start, stop;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start));

     
    hipStream_t stream1, stream2;
    checkKernelErrors(hipStreamCreate(&stream1)); 
    checkKernelErrors(hipStreamCreate(&stream2));

    int dev = findCudaDevice(argc, (const char **) argv);
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

    Parameters parameters();
      



    
    //TENSOR CORES PARAMETERS
    enum {
    //  // Compute the right amount of shared memory to request.
    // // We need shared memory to hold per-CTA C and D matrix tiles, and to cache
    // // per-CTA chunks
    // // of the A and B matrices. Therefore, the right amount to request is the
    // // maximum of those
    // // two numbers.
    SHMEM_SZ = MAX(
        sizeof(half) * (BLOCK_COL_TILES * M) * (CHUNK_K * K + SKEW_HALF) * 2,
        M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N *
           (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(half))
    }; 

   checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(
       compute_gemm), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ));

   checkKernelErrors(
        (compute_gemm<<<deviceProp.multiProcessorCount, THREADS_PER_BLOCK,
                      SHMEM_SZ, stream1>>>(a.data(), b.data(), c_h.data(), d_h.data(), half(1.0), half(0.0))));
   

       
    rad::checkFrameworkErrors(hipDeviceSynchronize());
    rad::checkFrameworkErrors(hipPeekAtLastError());
    
    // Device to host 
    d_h.to_vector(d_host);

    // write gold 
    if (GOLD){ 
        if (!write_to_file(d_host)){
          printf("not possible to write gold!\n");
        }else{
          printf("gold generated \n");
        }  
    } 


    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));

    float milliseconds = 0;

    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    printf("Kernel execution time: %f ms\n", milliseconds);   

    read_gold(gold_host);
    std::cout << "Starting the comparing process...\n";
    std::cout << std::setprecision(5) << std::fixed;

    auto errors = std::pair<int, int>();
    errors = check_output_errors_dmr<half,half>(gold_host, d_host,
            c_host, parameters, 0,
           DMR);

    std::cout << "#ERRORS" << errors << std::endl; 
    
    
        
  
}
